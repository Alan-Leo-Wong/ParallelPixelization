#include "hip/hip_runtime.h"
//
// Created by lei on 22-11-15.
//

#include <unistd.h>
#include "PointData.h"
#include "pixelData.h"
#include "utils/CUDA/cudaCheck.cuh"
#include "utils/common/timer.hpp"
#include "infoLog/infoPrint.hpp"
#include "utils/common/cmdLineParser.hpp"
#include "infoLog/log.hpp"
//#include "src/Visualization/MyViewer.h"

__device__ double d_end_w_coordinate;

__device__ double vector_product(const double *vectorA,
                                 const double *vectorB) {
    /*计算 x_1 * y_2 - x_2 * y_1*/
    return vectorA[0] * vectorB[1] - vectorB[0] * vectorA[1];
}

__device__ bool check_intersect(const double *A,
                                const double *B,
                                const double *C,
                                const double *D) {
    double AC[2] = {C[0] - A[0], C[1] - A[1]};
    double AD[2] = {D[0] - A[0], D[1] - A[1]};
    double BC[2] = {C[0] - B[0], C[1] - B[1]};
    double BD[2] = {D[0] - B[0], D[1] - B[1]};

    return (vector_product(AC, AD) * vector_product(BC, BD) <= 1e-15) &&
           (vector_product(AC, BC) * vector_product(AD, BD) <= 1e-15);
}

struct Point {
    double x, y;
};

struct Edge {
    int from, to;
};

__global__ void recasting_get_drawPixel(const Point *d_points_data,
                                        const Point *d_centers_data,
                                        const Edge *d_edges_index_data,
                                        int *d_draw_pixel_mesh,
                                        const int pixel_num, const int point_num) {

    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < pixel_num) {
        double x = d_centers_data[i].x;
        double y = d_centers_data[i].y;

        // 使用过像素中心点且平行于x轴的射线作为判断
        double line_point1[2] = {x, y};
        double line_point2[2] = {d_end_w_coordinate, y};

        int count = 0; // 与边的相交次数

        for (int j = 0; j < point_num; ++j) {
            // 边的两个端点
            double edge_point1[2] = {d_points_data[d_edges_index_data[j].from].x, // x1
                                     d_points_data[d_edges_index_data[j].from].y}; // y1
            double edge_point2[2] = {d_points_data[d_edges_index_data[j].to].x, // x2
                                     d_points_data[d_edges_index_data[j].to].y}; // y2

            if (check_intersect(line_point1, line_point2, edge_point1, edge_point2)) count++;
        }

        if (count & 1) atomicExch(&d_draw_pixel_mesh[i], 1);
    }
}

const char *default_file = "..//res//bigpolygon.wkt";

cmdLineParameter<char *> In("in");
cmdLineParameter<double> pixelSize("ps");
cmdLineParameter<int> threadsNum("num");
cmdLineReadable *params[] = {
        &In, &threadsNum, &pixelSize
};

int main(int argc, char **argv) {

    if (argc > 1) cmdLineParse(argc - 1, &argv[1], params);
    const char *filename = In.set ? In.value : default_file;
    LOG("file = ", std::string(filename), INFO);
    const double pixel_size = pixelSize.set ? pixelSize.value : 0.05;
    LOG("pixel_size = ", std::to_string(pixel_size), INFO);

    // prepare data on host
    PointData pointData = PointData(filename);
    Pixel pixel = Pixel(pixel_size, &pointData);
    std::vector<Eigen::MatrixXd> pixel_mesh = pixel.get_pixel_mesh();

    Eigen::MatrixXd points_data = pointData.get_points_data();
    int point_num = points_data.rows();
    Point *h_points = (Point *) malloc(sizeof(Point) * point_num);
    for (int i = 0; i < points_data.rows(); ++i) {
        h_points[i].x = points_data.row(i)[0];
        h_points[i].y = points_data.row(i)[1];
    }

    Eigen::MatrixXi edges_index_data = pointData.get_edges_index_data(); // 所有边的两个端点的索引
    Edge *h_edges_index = (Edge *) malloc(sizeof(Edge) * edges_index_data.rows());
    for (int i = 0; i < edges_index_data.rows(); ++i) {
        h_edges_index[i].from = edges_index_data.row(i)[0];
        h_edges_index[i].to = edges_index_data.row(i)[1];
    }

    double end_w_coordinate = pixel.get_end_w_coordinate();

    Eigen::MatrixXd centers_data = pixel.get_center();
    int pixel_num = centers_data.rows();
    LOG("pixel num = ", std::to_string(pixel_num), INFO);
    Point *h_centers = (Point *) malloc(sizeof(Point) * pixel_num);
    for (int i = 0; i < centers_data.rows(); ++i) {
        h_centers[i].x = centers_data.row(i)[0];
        h_centers[i].y = centers_data.row(i)[1];
    }

    int *h_draw_pixel_mesh = (int *) malloc(sizeof(int) * pixel_num); // 待绘制的像素网格

    // initialize device data
    Point *d_points;
    Point *d_centers;
    Edge *d_edges_index;
    int *d_draw_pixel_mesh;
    CUDA_CHECK(hipMalloc((void **) &d_points, sizeof(Point) * point_num));
    CUDA_CHECK(hipMalloc((void **) &d_centers, sizeof(Point) * pixel_num));
    CUDA_CHECK(hipMalloc((void **) &d_edges_index, sizeof(Edge) * edges_index_data.rows()));
    CUDA_CHECK(hipMalloc((void **) &d_draw_pixel_mesh, sizeof(int) * pixel_num));

    // copy data from host to device
    CUDA_CHECK(hipMemcpy(d_points, h_points, sizeof(Point) * point_num, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_centers, h_centers, sizeof(Point) * pixel_num, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_edges_index, h_edges_index, sizeof(Edge) * edges_index_data.rows(),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_end_w_coordinate), &end_w_coordinate, sizeof(double)));

    //
    const int block_size = 1024;
    LOG("block_size = ", std::to_string(block_size), INFO);
    const int block_i = (pixel_num + block_size - 1) / block_size;
    dim3 blockNum(block_i);
    dim3 threadsPerBlock(block_size);

    Timer timer("--------CUDA test--------");
    recasting_get_drawPixel<<<blockNum, threadsPerBlock>>>(d_points, d_centers, d_edges_index, d_draw_pixel_mesh,
                                                           pixel_num, point_num);
    hipDeviceSynchronize();
    timer.log();

    // copy result from device to host
    CUDA_CHECK(hipMemcpy(h_draw_pixel_mesh, d_draw_pixel_mesh, sizeof(int) * pixel_num, hipMemcpyDeviceToHost));

    // draw_pixel
    std::vector<Eigen::MatrixXd> draw_pixel_mesh;
    for (int i = 0; i < pixel_num; ++i) {
        if (h_draw_pixel_mesh[i]) {
            draw_pixel_mesh.emplace_back(pixel_mesh[i]);
//            std::cout << pixel_mesh[i] << std::endl;
        }
    }
    LOG("draw pixel nums = ", std::to_string(draw_pixel_mesh.size()), INFO);
    pixel.set_draw_pixel_mesh(draw_pixel_mesh);

    // free pointers on host
    free(h_points);
    free(h_centers);
    free(h_edges_index);
    free(h_draw_pixel_mesh);

    // free pointers on device
    CUDA_CHECK(hipFree(d_points));
    CUDA_CHECK(hipFree(d_edges_index));
    CUDA_CHECK(hipFree(d_centers));
    CUDA_CHECK(hipFree(d_draw_pixel_mesh));

    /*igl::opengl::glfw::MyViewer myViewer = igl::opengl::glfw::MyViewer(&pixel);
    Eigen::MatrixXd infoLog(4, 3);
    infoLog << 0.5, 0.5, 0,
            0.5, 0.5, 0,
            0.5, 0.5, 0,
            0.5, 0.5, 0;

    std::vector<Eigen::RowVector3d> color_vec;
    color_vec.emplace_back(Eigen::RowVector3d(1, 0, 0));
    color_vec.emplace_back(Eigen::RowVector3d(0, 1, 0));
    color_vec.emplace_back(Eigen::RowVector3d(0, 0, 1));
    std::vector<float> bb_point_size;
    bb_point_size.emplace_back(2.5);
    bb_point_size.emplace_back(1e-9);

    myViewer.set_data(color_vec);
//    myViewer.draw_point_and_bb(bb_point_size);
    myViewer.draw_pixel(infoLog);
    myViewer.launch();*/

    return EXIT_SUCCESS;
}